
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialWeightedClassNLLCriterion.cu"
#else

void THNN_(SpatialWeightedClassNLLCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           bool sizeAverage,
           THCTensor *weights,
	   THCTensor *spatialWeights,
           THCTensor *total_weight)
{
  THArgCheck(THCIndexTensor_(nDimension)(state, target) == 3, 1,
             "only batches of spatial targets supported (3D tensors)" \
             " but got targets of dimension: %d",
             THCIndexTensor_(nDimension)(state, target));
  THArgCheck(THCTensor_(nDimension)(state, input) == 4, 2,
             "only batches of spatial inputs supported (4D tensors), "      \
             "but got input of dimension: %d", THCTensor_(nDimension)(state, input));

  if (weights && THCTensor_(nElement)(state, weights) != THCTensor_(size)(state, input, 1)) {
    THError("weight tensor should be defined either for all or no classes");
  }

  if (weights)
    THCUNN_assertSameGPU(state, 6, input, target, weights, spatialWeights, output, total_weight); //5
  else
    THCUNN_assertSameGPU(state, 5, input, target, spatialWeights, output, total_weight); //4

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  spatialWeights = spatialWeights ? THCTensor_(newContiguous)(state, spatialWeights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *input_data = THCTensor_(data)(state, input);
  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  real *spatialWeights_data = spatialWeights ? THCTensor_(data)(state, spatialWeights) : NULL;
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  real *output_data = THCTensor_(data)(state, output);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  THCTensor_(fill)(state, output, ScalarConvert<int, real>::to(0));
  THCTensor_(fill)(state, total_weight, ScalarConvert<int, real>::to(0));

  cunn_SpatialWeightedClassNLLCriterion_updateOutput_kernel<real, accreal>
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      output_data,
      total_weight_data,
      input_data,
      target_data,
      weights_data,
      spatialWeights_data,
      sizeAverage,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) * THCTensor_(size)(state, input, 3),
      blocks_per_sample
  );
  THCudaCheck(cudaGetLastError());

  if (weights)
    THCTensor_(free)(state, weights);
  THCTensor_(free)(state, spatialWeights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

void THNN_(SpatialWeightedClassNLLCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *spatialWeights,
           THCTensor *total_weight)
{
  THArgCheck(THCIndexTensor_(nDimension)(state, target) == 3, 1,
             "only batches of spatial targets supported (3D tensors)");
  THArgCheck(THCTensor_(nDimension)(state, input) == 4, 2,
             "only batches of spatial inputs supported (4D tensors)");
  THArgCheck(THCTensor_(isContiguous)(state, gradInput), 4,
             "gradInput must be contiguous");
  if (weights && THCTensor_(nElement)(state, weights) != THCTensor_(size)(state, input, 1)) {
    THError("weight tensor should be defined either for all or no classes");
  }

  if (weights)
    THCUNN_assertSameGPU(state, 6, weights, spatialWeights, input, target, gradInput, total_weight); //5
  else
    THCUNN_assertSameGPU(state, 5, spatialWeights, input, target, gradInput, total_weight); //4

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  spatialWeights = spatialWeights ? THCTensor_(newContiguous)(state, spatialWeights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  real *spatialWeights_data = spatialWeights ? THCTensor_(data)(state, spatialWeights) : NULL;
  real *gradInput_data = THCTensor_(data)(state, gradInput);
  THCIndex_t *target_data = THCIndexTensor_(data)(state, target);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  cunn_SpatialWeightedClassNLLCriterion_updateGradInput_kernel
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      gradInput_data,
      target_data,
      weights_data,
      spatialWeights_data,
      total_weight_data,
      sizeAverage,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) *THCTensor_(size)(state, input, 3),
      blocks_per_sample
  );
  THCudaCheck(cudaGetLastError());

  if (weights)
    THCTensor_(free)(state, weights);
  
  THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

#endif
